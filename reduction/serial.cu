#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define BLOCK_SIZE 1024

__global__ void SerialKernel(const float* input, float* output, size_t n) {
    double sum = 0.0f;
    for (size_t i = 0; i < n; i++) {
        sum += input[i];
    }
    *output = sum;
}

void ReduceByKernel(const float* input, float* output, size_t n) {
    float* d_input;
    float* d_output;
    clock_t start = clock();
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, input, n * sizeof(float), hipMemcpyHostToDevice);
    clock_t start2 = clock();
    SerialKernel<<<1, 1>>>(d_input, d_output, n);
    clock_t end2 = clock();
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    clock_t end = clock();
    printf("ReduceByKernel\ncalculation time: %fus\n", (double)(end2 - start2) / CLOCKS_PER_SEC * 1000 * 1000);
    printf("I/O time: %fus\n", (double)(end - end2 + start2 - start) / CLOCKS_PER_SEC * 1000 * 1000);
    printf("Total time: %fus\n", (double)(end - start) / CLOCKS_PER_SEC * 1000 * 1000);
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void cudaGenRandomNumArray(float* array, int N, hiprandState* states){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N){
        hiprand_init(0, index, 0, &states[index]);
        array[index] = ((int)(hiprand_uniform(&states[index]) * 2.0)) / 8.0;
    }
}

void generate_array(float* array, int N){
    float* d_array;
    int block_test_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hiprandState* states;

    hipMalloc(&states, N * sizeof(hiprandState));
    hipMalloc(&d_array,N*sizeof(int));
    cudaGenRandomNumArray<<<block_test_num,1024>>>(d_array, N, states);
    hipMemcpy(array,d_array,N*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_array);
    hipFree(states);
}
